#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
    Recursive Gaussian filter
    sgreen 8/1/08

    This code sample implements a Gaussian blur using Deriche's recursive method:
    http://citeseer.ist.psu.edu/deriche93recursively.html

    This is similar to the box filter sample in the SDK, but it uses the previous
    outputs of the filter as well as the previous inputs. This is also known as an
    IIR (infinite impulse response) filter, since its response to an input impulse
    can last forever.

    The main advantage of this method is that the execution time is independent of
    the filter width.

    The GPU processes columns of the image in parallel. To avoid uncoalesced reads
    for the row pass we transpose the image and then transpose it back again
    afterwards.

    The implementation is based on code from the CImg library:
    http://cimg.sourceforge.net/
    Thanks to David Tschumperl� and all the CImg contributors!
*/

#include "cuda/cuda_recursive_gauss_filter.hh"

__global__
void d_recursiveGaussian_RGB(const PtrStep<uchar4> src, PtrStep<uchar4> dest,
                             int w, int h,
                             float a0, float a1, float a2, float a3,
                             float b1, float b2,
                             float coefp, float coefn)
{
   unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

   if (x >= w) return;

   // forward pass
   float4 xp = make_float4(0.0f);  // previous input
   float4 yp = make_float4(0.0f);  // previous output
   float4 yb = make_float4(0.0f);  // previous output by 2
#if CLAMP_TO_EDGE
   xp = gpuMatElemToFloat(src(0,x));
   yb = coefp*xp;
   yp = yb;
#endif

   for (int y = 0; y < h; y++)
   {
      float4 xc = gpuMatElemToFloat(src(y,x));
      float4 yc = a0*xc + a1*xp - b1*yp - b2*yb;
      dest(y,x) = floatToGpuMatElem(yc);
      xp = xc;
      yb = yp;
      yp = yc;
   }

   // reverse pass
   // ensures response is symmetrical
   float4 xn = make_float4(0.0f);
   float4 xa = make_float4(0.0f);
   float4 yn = make_float4(0.0f);
   float4 ya = make_float4(0.0f);
#if CLAMP_TO_EDGE
   xn = xa = gpuMatElemToFloat(src(h-1,x));
   yn = coefn*xn;
   ya = yn;
#endif

   for (int y = h-1; y >= 0; y--)
   {
      float4 xc = gpuMatElemToFloat(src(y,x));
      float4 yc = a2*xn + a3*xa - b1*yn - b2*ya;
      xa = xn;
      xn = xc;
      ya = yn;
      yn = yc;
      dest(y,x) = floatToGpuMatElem(gpuMatElemToFloat(dest(y,x)) + yc);
   }
}

void gaussianFilter_RGB_GPU(const GpuMat& d_src, GpuMat& d_dest, GpuMat& d_temp,
                            float sigma, int order, int nthreads)
{
   int width = d_src.cols;
   int height = d_src.rows;

   //Make sure GpuMats have the same dimensions
   d_dest.cols = d_temp.cols = width;
   d_dest.rows = d_temp.rows = height;
   d_dest.step = d_temp.step = d_src.step;
   
   //Compute filter coefficients
   const float
      nsigma = sigma < 0.1f ? 0.1f : sigma,
   		       alpha = 1.695f / nsigma,
   		       ema = (float)std::exp(-alpha),
   		       ema2 = (float)std::exp(-2*alpha),
   		       b1 = -2*ema,
   		       b2 = ema2;

   float a0 = 0, a1 = 0, a2 = 0, a3 = 0, coefp = 0, coefn = 0;

   switch (order)
   {
      case 0:
      {
   	 const float k = (1-ema)*(1-ema)/(1+2*alpha*ema-ema2);
   	 a0 = k;
   	 a1 = k*(alpha-1)*ema;
   	 a2 = k*(alpha+1)*ema;
   	 a3 = -k*ema2;
      }
      break;

      case 1:
      {
   	 const float k = (1-ema)*(1-ema)/ema;
   	 a0 = k*ema;
   	 a1 = a3 = 0;
   	 a2 = -a0;
      }
      break;

      case 2:
      {
   	 const float
   	    ea = (float)std::exp(-alpha),
   	    k = -(ema2-1)/(2*alpha*ema),
   	    kn = (-2*(-1+3*ea-3*ea*ea+ea*ea*ea)/(3*ea+1+3*ea*ea+ea*ea*ea));
   	 a0 = kn;
   	 a1 = -kn*(1+k*alpha)*ema;
   	 a2 = kn*(1-k*alpha)*ema;
   	 a3 = -kn*ema2;
      }
      break;

      default:
   	 fprintf(stderr, "gaussianFilter: invalid order parameter!\n");
   	 return;
   }

   coefp = (a0+a1)/(1+b1+b2);
   coefn = (a2+a3)/(1+b1+b2);
   
   d_recursiveGaussian_RGB<<< iDivUp(width, nthreads), nthreads >>>(d_src,
   								     d_temp,
   								     width,
   								     height,
   								     a0, a1, a2, a3,
   								     b1, b2,
   								     coefp,
   								     coefn);
   getLastCudaError("Kernel execution failed");
   
   transpose_GPU(d_temp, d_dest);
   getLastCudaError("transpose: Kernel execution failed");

   //Adjust temp dimensions to dest dimensions for recursive gaussian pass
   d_temp.rows = d_dest.rows;
   d_temp.cols = d_dest.cols;
   d_temp.step = d_dest.step;
   
   d_recursiveGaussian_RGB<<< iDivUp(height, nthreads), nthreads >>>(d_dest,
   								      d_temp,
   								      height,
   								      width,
   								      a0, a1, a2, a3,
   								      b1, b2,
   								      coefp,
   								      coefn);

   getLastCudaError("Kernel execution failed");

   transpose_GPU(d_temp, d_dest);
}
